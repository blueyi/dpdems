﻿/*
 * thread_block_test.cu
 * Copyright (C) 2016  <@A0835-PC>
 *
 * Distributed under terms of the MIT license.
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

int N = 21504;

int blockPerGrid(const int dim, const int threadPerBlock)
{
   int temp = dim / threadPerBlock;
   if (dim % threadPerBlock != 0) {
      temp += 1; 
   }
   return temp;
}

__device__ void initial_a(int *a, int tid)
{
    a[tid] = tid * tid;
}

__device__ void initial_b(int *b)
{
    int tid = threadIdx.x;
    b[tid] = -tid;
}
__global__ void add(long long *a, long long *b, long long *c, int *bdim, int *gdim)
{
//    int tid = blockIdx.x;
    long long tid = threadIdx.x + blockIdx.x * blockDim.x;
//    while (tid < N) {
        a[tid] = tid * tid;
        b[tid] = -tid;
        c[tid] = a[tid] + b[tid];
//        tid += blockDim.x * gridDim.x;
//    }
    if (tid == 0) {
        *bdim = blockDim.x;
        *gdim = gridDim.x;
    }
}

inline void checkCudaError(hipError_t error, const char *file, const int line)
{
   if (error != hipSuccess) {
      std::cerr << "CUDA CALL FAILED: " << file << "( " << line << " )- " <<
         hipGetErrorString(error) << std::endl;
      exit(EXIT_FAILURE);
   }
   else
      std::cout << "cuda call success" << std::endl;
}

inline void checkCudaState(const char *msg, const char *file, const int line)
{
   hipError_t error = hipGetLastError();
   if (error != hipSuccess) {
      std::cerr << "---" << msg << " Error--" << std::endl;
      std::cerr << file << "( " << line << " )- " << 
         hipGetErrorString(error) << std::endl;
      exit(EXIT_FAILURE);
   }
   else
      std::cout << "cuda state Success: " << msg << std::endl;
}

#define CHECK_ERROR(error) checkCudaError(error, __FILE__, __LINE__);
#define CHECK_STATE(msg) checkCudaState(msg, __FILE__, __LINE__);

void print_device(const int id)
{
   hipDeviceProp_t props;
   CHECK_ERROR(hipGetDeviceProperties(&props, id));
   std::cout << "---Property of currently device used---" << std::endl;
   std::cout << "Device " << id << ": " << props.name << std::endl;
   std::cout << "CUDA Capability: " << props.major << "." << props.minor
      << std::endl;
   std::cout << "MultiProcessor count: " << props.multiProcessorCount << std::endl;
}

void setCudaDevice(int id)
{
   int numDevice = 0;
   CHECK_ERROR(hipGetDeviceCount(&numDevice));
   std::cout << "Total CUDA device number: " << numDevice << std::endl;
   if (numDevice > 1) {
      hipDeviceProp_t props;
      hipGetDeviceProperties(&props, id);
      int maxMultiProcessors = props.multiProcessorCount;
      for (int device = 1; device < numDevice; ++device) {
         CHECK_ERROR(hipGetDeviceProperties(&props, device));
         if (maxMultiProcessors < props.multiProcessorCount) {
            maxMultiProcessors = props.multiProcessorCount;
            id = device;
         }
      }
   }
   CHECK_ERROR(hipSetDevice(id));
   print_device(id);
}


int main(int argc, char **argv)
{
   int id = 0;
   setCudaDevice(id);
   if (argc > 1)
      N = atoi(argv[1]);
   long long *c = new long long[N];
   int bdim, gdim;
   long long *dev_a, *dev_b, *dev_c;
   int *dev_bdim, *dev_gdim;
   hipMalloc(&dev_a, N * sizeof(long long));
   hipMalloc(&dev_b, N * sizeof(long long));
   hipMalloc(&dev_c, N * sizeof(long long));
   hipMalloc(&dev_bdim, sizeof(int));
   hipMalloc(&dev_gdim, sizeof(int));
   //    add<<<N, 1>>>(dev_a, dev_b, dev_c);
   int threadPerBlock = 256;
   int blockSize = blockPerGrid(N, threadPerBlock);
   add<<<blockSize, threadPerBlock>>>(dev_a, dev_b, dev_c, dev_bdim, dev_gdim);
   hipMemcpy(c, dev_c, N * sizeof(long long), hipMemcpyDeviceToHost);
   hipMemcpy(&bdim, dev_bdim, sizeof(int), hipMemcpyDeviceToHost);
   hipMemcpy(&gdim, dev_gdim, sizeof(int), hipMemcpyDeviceToHost);
   hipFree(dev_a);
   hipFree(dev_b);
   hipFree(dev_c);
   hipFree(dev_bdim);
   hipFree(dev_gdim);
   for (int i = 0; i < N; ++i) {
      std::cout << i << ": " << c[i] << std::endl; 
   }
   std::cout << "blockDim.x: " << bdim << std::endl;
   std::cout << "gridDim.x: " << gdim << std::endl;
   free(c);
   return 0;
}


