#include "hip/hip_runtime.h"
/*
 * testCuda.cu
 * Copyright (C) 2016  <@BLUEYI-PC>
 *
 * Distributed under terms of the MIT license.
 */

#include "../include/common.h"
#include <iostream>

__global__ void add(XYZ<double> *a, XYZ<double> *b, XYZ<int> *c)
{
    c->x = a->x + b->x;
    c->y = a->y + b->y;
    c->z = a->z + b->z;
}

int main(void)
{
    XYZ<double> a(1.0, 2.0, 3.0);
    XYZ<double> b(1.0, 2.0, 3.0);
    XYZ<int> c;
    XYZ<double> *dev_a;
    XYZ<double> *dev_b;
    XYZ<int> *dev_c;
    hipMalloc((void**)&dev_a, sizeof(a));
    hipMalloc((void**)&dev_b, sizeof(b));
    hipMalloc((void**)&dev_c, sizeof(c));
    hipMemcpy(dev_a, &a, sizeof(a), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, &b, sizeof(b), hipMemcpyHostToDevice);
    add<<<1, 1>>>(dev_a, dev_b, dev_c);
    hipMemcpy(&c, &dev_c, sizeof(c), hipMemcpyDeviceToHost);
    c.print(std::cout);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}


