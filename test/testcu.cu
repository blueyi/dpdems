/*
 * testCuda.cu
 * Copyright (C) 2016  <@BLUEYI-PC>
 *
 * Distributed under terms of the MIT license.
 */


#include <hip/hip_runtime.h>
#include <iostream>
struct XYZ {
    int x;
    int y;
    int z;
};

__global__ void add(XYZ *a, XYZ *b, XYZ *c)
{
    c->x = a->x + b->x;
    c->y = a->y + b->y;
    c->z = a->z + b->z;
}

int main(void)
{
    XYZ a{1, 2, 3};
    XYZ b{1, 2, 3};
    XYZ c;
    XYZ *dev_a;
    XYZ *dev_b;
    XYZ *dev_c;
    hipMalloc((void**)&dev_a, sizeof(a));
    hipMalloc((void**)&dev_b, sizeof(b));
    hipMalloc((void**)&dev_c, sizeof(c));
    hipMemcpy(dev_a, &a, sizeof(a), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, &b, sizeof(b), hipMemcpyHostToDevice);
    add<<<1, 1>>>(dev_a, dev_b, dev_c);
    hipMemcpy(&c, &dev_c, sizeof(c), hipMemcpyDeviceToHost);
    std::cout << c.x << " " << c.y << " " << c.z << std::endl;
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}


