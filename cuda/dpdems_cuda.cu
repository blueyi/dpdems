#include "hip/hip_runtime.h"
/*
 * dpdems_cuda.cu
 * Copyright (C) 2016  <@A0835-PC>
 *
 * Distributed under terms of the MIT license.
 */

#include "..\include\inputDatas.h"
#include "..\include\config.h"
#include <iostream>
#include <fstream>
#include <cctype>
#include <ctime>

const int maxThreads = 21504;
const int threadPerBlock = 512;
int blockPerGrid(const int dim, const int threadPerBlock)
{
    return (dim + threadPerBlock - 1) / threadPerBlock;
}

inline void checkCudaError(hipError_t error, const char *file, const int line)
{
    if (error != hipSuccess) {
        std::cerr << "CUDA CALL FAILED: " << file << "( " << line << " )- " <<
            hipGetErrorString(error) << std::endl;
        exit(EXIT_FAILURE);
    }
}

inline void checkCudaState(const char *msg, const char *file, const int line)
{
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "---" << msg << " Error--" << std::endl;
        std::cerr << file << "( " << line << " )- " << 
            hipGetErrorString(error) << std::endl;
        exit(EXIT_FAILURE);
    }
}

#define CHECK_ERROR(error) checkCudaError(error, __FILE__, __LINE__);
#define CHECK_STATE(msg) checkCudaState(msg, __FILE__, __LINE__);

void print_device(const int id)
{
    hipDeviceProp_t props;
    CHECK_ERROR(hipGetDeviceProperties(&props, id));
    std::cout << "---Property of currently device used---" << std::endl;
    std::cout << "Device " << id << ": " << props.name << std::endl;
    std::cout << "CUDA Capability: " << props.major << "." << props.minor
        << std::endl;
    std::cout << "MultiProcessor count: " << props.multiProcessorCount << std::endl;
}

void setCudaDevice(int id)
{
    int numDevice = 0;
    CHECK_ERROR(hipGetDeviceCount(&numDevice));
    std::cout << "Total CUDA device number: " << numDevice << std::endl;
    if (numDevice > 1) {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, id);
        int maxMultiProcessors = props.multiProcessorCount;
        for (int device = 1; device < numDevice; ++device) {
            CHECK_ERROR(hipGetDeviceProperties(&props, device));
            if (maxMultiProcessors < props.multiProcessorCount) {
                maxMultiProcessors = props.multiProcessorCount;
                id = device;
            }
        }
    }
    CHECK_ERROR(hipSetDevice(id));
    print_device(id);
}

void init(std::vector<double *>&, const std::vector<Particle>&);

//unsigned hit(std::vector<ParticlePtr> &ppv, Grid &grid, unsigned long time, std::ostream &os, const std::vector<Particle> &pv);

int main(int argc, char **argv)
{
   std::cout.setf(std::ios::scientific);
   std::cout.precision(19);
   std::string configFile = "config.txt";

   if (ini_conf(configFile.c_str()))
      std::cout << "Success" << std::endl;
   else 
      std::cout << "ini_conf error!" << std::endl;

   std::string ifileName = dataFile;
   std::ifstream inf;
   if (2 > argc){
      std::cout << "Use the default input file name from config.txt: inputdatas.txt" << std::endl;
   }
   else {
      ifileName = argv[1];
      std::cout << "Use input file: " << ifileName << std::endl;
   }
   inf.open(ifileName);
   if (!inf) {
      std::cout << "File Error: " << ifileName << std::endl;
      return 0;
   }
   unsigned particle_num = 0;
   double ttime0, dt, elasticmod, poissonp, rho, xlen, ylen, zlen;
   inf >> particle_num >> ttime0 >> dt >> elasticmod >>
      poissonp >> rho >> xlen >> ylen >> zlen;

   if ( 3 > maxdim) {
      std::cout << "maxdim too small" << std::endl;
      std::cout << "Execute terminate!" << std::endl;
      return 0;
   }

   if (10000000 < timestep * stepnum) {
      std::string str;
      std::cout << "timestep * stepnum too big, it may excute too long, contine? Y/N: " << std::endl;
      std::cin>> str;
      for (auto &c : str) {
         c = tolower(c);
      }
      if (str[0] != 'y') {
         std::cout << "Execute terminate!" << std::endl;
         return 0;
      }
   }

   if (maxdim >= 300) {
      std::string str;
      std::cout << "You need at least 1.7G memory, contine? Y/N: " << std::endl;
      std::cin>> str;
      for (auto &c : str) {
         c = tolower(c);
      }
      if (str[0] != 'y') {
         std::cout << "Execute terminate!" << std::endl;
         return 0;
      }

      if (maxdim >= 500) {
         std::string str;
         std::cout << "You need at least 6.5G memory, contine? Y/N: " << std::endl;
         std::cin>> str;
         for (auto &c : str) {
            c = tolower(c);
         }
         if (str[0] != 'y') {
            std::cout << "Execute terminate!" << std::endl;
            return 0;
         }
      }

   }

   std::string ofs_result = ifileName + ".log";
   std::ofstream ofresult(ofs_result);

   std::cout << " Particle Num: " << particle_num << std::endl;
   std::cout << "    Time step: " << timestep << std::endl;
   std::cout << "      Max dim: -" << maxdim << " ~ +" << maxdim << std::endl;
   std::cout << "Time step num: " << stepnum << std::endl;
   std::cout << std::endl << "************Start*************" << std::endl;

   ofresult << " Particle Num: " << particle_num << std::endl;
   ofresult << "    Time step: " << timestep << std::endl;
   ofresult << "      Max dim: -" << maxdim << " ~ +" << maxdim << std::endl;
   ofresult << "Time step num: " << stepnum << std::endl;
   ofresult << std::endl << "************Start*************" << std::endl;



//   clock_t t;
//   t = clock();

   std::vector<Particle> pv(particle_num);
   std::size_t readnum = 0;
   for (auto &p : pv) {
      if (!inf)
         break;
      ++readnum;
      p.asign(inf);
   }
   std::cout << "Read particle data num: " << readnum << std::endl << std::endl;
   ofresult << "Read particle data num: " << readnum << std::endl << std::endl;
   inf.close();
   particle_num = readnum;
   pv.resize(particle_num);



   double maxx, maxy, maxz;
   maxx = fabs((pv[0]).xyz.x);
   maxy = fabs((pv[0]).xyz.y);
   maxz = fabs((pv[0]).xyz.z);
   for (auto p : pv) {
      if (maxx < fabs(p.xyz.x))
         maxx = fabs(p.xyz.x);
      if (maxy < fabs(p.xyz.y))
         maxy = fabs(p.xyz.y);
      if (maxz < fabs(p.xyz.z))
         maxz = fabs(p.xyz.z);
   }

   int grid_maxx, grid_maxy, grid_maxz;
   grid_maxx = grid_maxy = grid_maxz = maxdim;

   XYZ<int> grid_limit(grid_maxx - 1, grid_maxy - 1, grid_maxz - 1);
   int scal_x, scal_y, scal_z;
   scal_x = maxx == 0.0 ? 0.0 : (double)grid_limit.x / maxx;
   scal_y = maxy == 0.0 ? 0.0 : (double)grid_limit.y / maxy;
   scal_z = maxz == 0.0 ? 0.0 : (double)grid_limit.z / maxz;

   double *xxt = new double(readnum + 1);
   double *xyt = new double(readnum + 1);
   double *xzt = new double(readnum + 1);
   double *vx = new double(readnum);
   double *vy = new double(readnum);
   double *vz = new double(readnum);
   std::vector<double *> ppv{xxt, xyt, xzt, vx, vy, vz};
   init(ppv, pv);
   xxt[readnum] = scal_x;
   xyt[readnum] = scal_y;
   xzt[readnum] = scal_z;
   std::cout << xxt[0] << std::endl;
   std::cout << vz[0] << std::endl;

   int *x = new int(readnum);
   int *y = new int(readnum);
   int *z = new int(readnum);
   int *dev_x;
   int *dev_y;
   int *dev_z;
   int *dev_xt;
   int *dev_yt;
   int *dev_zt;
   hipMalloc((void**)&dev_x, readnum * sizeof(int));
   hipMalloc((void**)&dev_y, readnum * sizeof(int));
   hipMalloc((void**)&dev_z, readnum * sizeof(int));
   hipMalloc((void**)&dev_xt, (readnum + 1) * sizeof(int));
   hipMalloc((void**)&dev_yt, (readnum + 1) * sizeof(int));
   hipMalloc((void**)&dev_zt, (readnum + 1) * sizeof(int));
   
   

/*
   //   for (auto pp : ppv) {
   //      std::cout << "*" << pp.no() << "*" << std::endl;
   //      pp.print(std::cout);
   //   }

   std::size_t gdimx = axis_conv(grid_maxx, abs(grid_minx));
   std::size_t gdimy = axis_conv(grid_maxy, abs(grid_miny));
   std::size_t gdimz = axis_conv(grid_maxz, abs(grid_minz));

   XYZ<int> offset(grid_maxx, grid_maxy, grid_maxz);
   Grid grid(gdimx, gdimy, gdimz, offset);
   grid.fill(ppv);




//   hit(ppv, grid, timestep * stepnum, ofresult, pv);
//   t = clock() - t;
   double seconds = (double)t / CLOCKS_PER_SEC;

   std::cout << std::endl << "Total time consumed: " << seconds << " seconds" << std::endl;
   std::cout << "Result output to file: " << ofs_result << std::endl;

   std::cout << std::endl << "************Config Info*************" << std::endl;
   std::cout << " Particle Num: " << particle_num << std::endl;
   std::cout << "    Time step: " << timestep << std::endl;
   std::cout << "      Max dim: -" << maxdim << " ~ +" << maxdim << std::endl;
   std::cout << "Time step num: " << stepnum << std::endl;
   std::cout << std::endl << "************End*************" << std::endl;

   ofresult << std::endl << "Total time consumed: " << seconds << " seconds" << std::endl;
   ofresult << std::endl << "************Config Info*************" << std::endl;
   ofresult << " Particle Num: " << particle_num << std::endl;
   ofresult << "    Time step: " << timestep << std::endl;
   ofresult << "      Max dim: -" << maxdim << " ~ +" << maxdim << std::endl;
   ofresult << "Time step num: " << stepnum << std::endl;
   ofresult << std::endl << "************End*************" << std::endl;

   ofresult.close();

   //   for (auto pp : ppv) {
   //      std::cout << "*" << pp.no() << "*" << std::endl;
   //      pp.print(std::cout);
   //   }

   //   std::cout << "Unnull: " << grid.unNullPtrNum() << std::endl;

   //std::cout << maxx << " " << maxy << " " << maxz << std::endl;
   //std::cout << p0.x << " " << p0.y << " " << p0.z << std::endl;
   //std::cout << (pv[0]).xyz.x << " " << (pv[0]).xyz.y << " " << (pv[0]).xyz.z;
   std::cout << std::endl;
   */
   return 0;
}

void init(std::vector<double *> &ppv, const std::vector<Particle> &pv)
{
    int readnum = pv.size();
    for (int i = 0; i < readnum; ++i) {
        *(ppv[0] + i) = (pv[i]).xyz.x;
        *(ppv[1] + i) = (pv[i]).xyz.y;
        *(ppv[2] + i) = (pv[i]).xyz.z;
        *(ppv[3] + i) = (pv[i]).v.x;
        *(ppv[4] + i) = (pv[i]).v.y;
        *(ppv[5] + i) = (pv[i]).v.z;
    }
}

/*
unsigned hit(std::vector<ParticlePtr> &ppv, Grid &grid, unsigned long time, std::ostream &os, const std::vector<Particle> &pv)
{
    unsigned long long total_hit = 0;
    for (auto &pp : ppv) {
        unsigned hit_times = pp.move(grid, time);
        total_hit += hit_times;
        std::cout << std::endl << "Particle " << pp.no() << " hit times: " << hit_times << std::endl;
        std::cout << "      Total hit times: " << total_hit << std::endl;
        std::cout << "Particle current info: " << std::endl;
        pp.print(std::cout);

        os.setf(std::ios::scientific);
        os.precision(19);
        os << std::endl << "********************" << std::endl;
        os << "Particle " << pp.no() << " hit times: " << hit_times << std::endl;
        os << "Particle origin info: " << std::endl;
        (pv[pp.no() - 1]).print(os);
        os << "Particle current info: " << std::endl;
        pp.print(os);
        os << "Total hit times: " << total_hit << std::endl << std::endl;
    }
    return total_hit;
}
*/
