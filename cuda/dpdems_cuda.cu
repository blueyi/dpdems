#include "hip/hip_runtime.h"
/*
 * dpdems_cuda.cu
 * Copyright (C) 2016  <@A0835-PC>
 *
 * Distributed under terms of the MIT license.
 */

#include "..\include\inputDatas.h"
#include "..\include\config.h"
#include <iostream>
#include <fstream>
#include <cctype>
#include <ctime>
#include "hip/hip_runtime.h"
#include ""


const int maxThreads = 21504;
const int threadPerBlock = 512;
int blockPerGrid(const int dim, const int threadPerBlock)
{
   return (dim + threadPerBlock - 1) / threadPerBlock;
}

inline void checkCudaError(hipError_t error, const char *file, const int line)
{
   if (error != hipSuccess) {
      std::cerr << "CUDA CALL FAILED: " << file << "( " << line << " )- " <<
         hipGetErrorString(error) << std::endl;
      exit(EXIT_FAILURE);
   }
   else
      std::cout << "cuda call success" << std::endl;
}

inline void checkCudaState(const char *msg, const char *file, const int line)
{
   hipError_t error = hipGetLastError();
   if (error != hipSuccess) {
      std::cerr << "---" << msg << " Error--" << std::endl;
      std::cerr << file << "( " << line << " )- " << 
         hipGetErrorString(error) << std::endl;
      exit(EXIT_FAILURE);
   }
   else
      std::cout << "cuda state Success: " << msg << std::endl;
}

#define CHECK_ERROR(error) checkCudaError(error, __FILE__, __LINE__);
#define CHECK_STATE(msg) checkCudaState(msg, __FILE__, __LINE__);

void print_device(const int id)
{
   hipDeviceProp_t props;
   CHECK_ERROR(hipGetDeviceProperties(&props, id));
   std::cout << "---Property of currently device used---" << std::endl;
   std::cout << "Device " << id << ": " << props.name << std::endl;
   std::cout << "CUDA Capability: " << props.major << "." << props.minor
      << std::endl;
   std::cout << "MultiProcessor count: " << props.multiProcessorCount << std::endl;
}

void setCudaDevice(int id)
{
   int numDevice = 0;
   CHECK_ERROR(hipGetDeviceCount(&numDevice));
   std::cout << "Total CUDA device number: " << numDevice << std::endl;
   if (numDevice > 1) {
      hipDeviceProp_t props;
      hipGetDeviceProperties(&props, id);
      int maxMultiProcessors = props.multiProcessorCount;
      for (int device = 1; device < numDevice; ++device) {
         CHECK_ERROR(hipGetDeviceProperties(&props, device));
         if (maxMultiProcessors < props.multiProcessorCount) {
            maxMultiProcessors = props.multiProcessorCount;
            id = device;
         }
      }
   }
   CHECK_ERROR(hipSetDevice(id));
   print_device(id);
}

void init(std::vector<double *>&, const std::vector<Particle>&);

__global__ void cudaScale(double *dev_xt, double *dev_yt, double *dev_zt, unsigned *dev_x, unsigned *dev_y, unsigned *dev_z, int readnum, int maxdim)
{
   int tid = threadIdx.x + blockIdx.x * blockDim.x;
   while (tid < readnum) {
      dev_x[tid] = dev_xt[tid] * dev_xt[readnum] + maxdim;
      dev_y[tid] = dev_yt[tid] * dev_yt[readnum] + maxdim;
      dev_z[tid] = dev_zt[tid] * dev_zt[readnum] + maxdim;
      tid += blockDim.x * gridDim.x;
   }
}

double scalev(double &, const double &);
void swapv(double *, double *, double *, int, int, double);
bool isInGrid(const int &, const int &, const int &, const int &);
unsigned updatePosition(unsigned *, unsigned *, unsigned *, double *, double *, double *, const int &, const int &, int ***, const unsigned long &);
unsigned long long collision(unsigned *, unsigned *, unsigned *, double *, double *, double *, const int &, const int &, int ***, const unsigned long &, std::ostream &);

int main(int argc, char **argv)
{
   std::cout.setf(std::ios::scientific);
   std::cout.precision(19);
   std::string configFile = "config.txt";

   if (ini_conf(configFile.c_str()))
      std::cout << "Success" << std::endl;
   else 
      std::cout << "ini_conf error!" << std::endl;

   std::string ifileName = dataFile;
   std::ifstream inf;
   if (2 > argc){
      std::cout << "Use the default input file name from config.txt: inputdatas.txt" << std::endl;
   }
   else {
      ifileName = argv[1];
      std::cout << "Use input file: " << ifileName << std::endl;
   }
   inf.open(ifileName);
   if (!inf) {
      std::cout << "File Error: " << ifileName << std::endl;
      return 0;
   }
   unsigned particle_num = 0;
   double ttime0, dt, elasticmod, poissonp, rho, xlen, ylen, zlen;
   inf >> particle_num >> ttime0 >> dt >> elasticmod >>
      poissonp >> rho >> xlen >> ylen >> zlen;

   if ( 3 > maxdim) {
      std::cout << "maxdim too small" << std::endl;
      std::cout << "Execute terminate!" << std::endl;
      return 0;
   }

   if (10000000 < timestep * stepnum) {
      std::string str;
      std::cout << "timestep * stepnum too big, it may excute too long, contine? Y/N: " << std::endl;
      std::cin>> str;
      for (auto &c : str) {
         c = tolower(c);
      }
      if (str[0] != 'y') {
         std::cout << "Execute terminate!" << std::endl;
         return 0;
      }
   }

   if (maxdim >= 300) {
      std::string str;
      std::cout << "You need at least 1.7G memory, contine? Y/N: " << std::endl;
      std::cin>> str;
      for (auto &c : str) {
         c = tolower(c);
      }
      if (str[0] != 'y') {
         std::cout << "Execute terminate!" << std::endl;
         return 0;
      }

      if (maxdim >= 500) {
         std::string str;
         std::cout << "You need at least 6.5G memory, contine? Y/N: " << std::endl;
         std::cin>> str;
         for (auto &c : str) {
            c = tolower(c);
         }
         if (str[0] != 'y') {
            std::cout << "Execute terminate!" << std::endl;
            return 0;
         }
      }

   }

   std::string ofs_result = ifileName + ".log";
   std::ofstream ofresult(ofs_result);

   std::cout << " Particle Num: " << particle_num << std::endl;
   std::cout << "    Time step: " << timestep << std::endl;
   std::cout << "      Max dim: -" << maxdim << " ~ +" << maxdim << std::endl;
   std::cout << "Time step num: " << stepnum << std::endl;
   std::cout << std::endl << "************Start*************" << std::endl;

   ofresult << " Particle Num: " << particle_num << std::endl;
   ofresult << "    Time step: " << timestep << std::endl;
   ofresult << "      Max dim: -" << maxdim << " ~ +" << maxdim << std::endl;
   ofresult << "Time step num: " << stepnum << std::endl;
   ofresult << std::endl << "************Start*************" << std::endl;



   std::vector<Particle> pv(particle_num);
   std::size_t readnum = 0;
   for (auto &p : pv) {
      if (!inf)
         break;
      ++readnum;
      p.asign(inf);
   }
   std::cout << "Read particle data num: " << readnum << std::endl << std::endl;
   ofresult << "Read particle data num: " << readnum << std::endl << std::endl;
   inf.close();
   particle_num = readnum;
   pv.resize(particle_num);

   int device_id = 0;
   setCudaDevice(device_id);

   double maxx, maxy, maxz;
   maxx = fabs((pv[0]).xyz.x);
   maxy = fabs((pv[0]).xyz.y);
   maxz = fabs((pv[0]).xyz.z);
   for (auto p : pv) {
      if (maxx < fabs(p.xyz.x))
         maxx = fabs(p.xyz.x);
      if (maxy < fabs(p.xyz.y))
         maxy = fabs(p.xyz.y);
      if (maxz < fabs(p.xyz.z))
         maxz = fabs(p.xyz.z);
   }

   int grid_maxx, grid_maxy, grid_maxz;
   grid_maxx = grid_maxy = grid_maxz = maxdim;

   XYZ<int> grid_limit(grid_maxx - 1, grid_maxy - 1, grid_maxz - 1);
   double scal_x, scal_y, scal_z;
   scal_x = maxx == 0.0 ? 0.0 : (double)grid_limit.x / maxx;
   scal_y = maxy == 0.0 ? 0.0 : (double)grid_limit.y / maxy;
   scal_z = maxz == 0.0 ? 0.0 : (double)grid_limit.z / maxz;

   double *xt = new double(readnum + 1);
   double *yt = new double(readnum + 1);
   double *zt = new double(readnum + 1);
   double *vx = new double(readnum);
   double *vy = new double(readnum);
   double *vz = new double(readnum);
   std::vector<double *> ppvt{xt, yt, zt, vx, vy, vz};
   init(ppvt, pv);
   xt[readnum] = scal_x;
   yt[readnum] = scal_y;
   zt[readnum] = scal_z;

   std::cout << vz[0] << std::endl;
   std::cout << xt[0] << std::endl;

   clock_t t;
   t = clock();

   hipEvent_t start, stop;
   CHECK_ERROR(hipEventCreate(&start));
   CHECK_ERROR(hipEventCreate(&stop));
   CHECK_ERROR(hipEventRecord(start, 0));
   CHECK_ERROR(hipEventSynchronize(start));

   unsigned *x = new unsigned(readnum);
   unsigned *y = new unsigned(readnum);
   unsigned *z = new unsigned(readnum);
   unsigned *dev_x;
   unsigned *dev_y;
   unsigned *dev_z;
   CHECK_STATE("debug1");
   CHECK_ERROR(hipMalloc((void**)&dev_x, readnum * sizeof(unsigned)));
   CHECK_STATE("debug2");

   CHECK_ERROR(hipMalloc((void**)&dev_y, readnum * sizeof(unsigned)));
   CHECK_ERROR(hipMalloc((void**)&dev_z, readnum * sizeof(unsigned)));

   double *dev_xt;
   double *dev_yt;
   double *dev_zt;

   CHECK_STATE("debug1");
   CHECK_ERROR(hipMalloc((void**)&dev_xt, (readnum + 1) * sizeof(double)));
   CHECK_STATE("debug2");
   CHECK_ERROR(hipMalloc((void**)&dev_yt, (readnum + 1) * sizeof(double)));
   CHECK_ERROR(hipMalloc((void**)&dev_zt, (readnum + 1) * sizeof(double)));

   CHECK_ERROR(hipMemcpy(dev_xt, xt, (readnum + 1) * sizeof(double), hipMemcpyHostToDevice));
   CHECK_ERROR(hipMemcpy(dev_yt, yt, (readnum + 1) * sizeof(double), hipMemcpyHostToDevice));
   CHECK_ERROR(hipMemcpy(dev_zt, zt, (readnum + 1) * sizeof(double), hipMemcpyHostToDevice));
   int threads = threadPerBlock;
   int blocks = blockPerGrid(readnum, threads);
   cudaScale<<<blocks, threads>>>(dev_xt, dev_yt, dev_zt, dev_x, dev_y, dev_z, readnum, maxdim);
   CHECK_STATE("cudaScale call");
   CHECK_ERROR(hipMemcpy(x, dev_x, readnum * sizeof(unsigned), hipMemcpyDeviceToHost));
   CHECK_ERROR(hipMemcpy(y, dev_y, readnum * sizeof(unsigned), hipMemcpyDeviceToHost));
   CHECK_ERROR(hipMemcpy(z, dev_z, readnum * sizeof(unsigned), hipMemcpyDeviceToHost));
   CHECK_ERROR(hipFree(dev_x));
   CHECK_ERROR(hipFree(dev_y));
   CHECK_ERROR(hipFree(dev_z));
   CHECK_ERROR(hipFree(dev_xt));
   CHECK_ERROR(hipFree(dev_yt));
   CHECK_ERROR(hipFree(dev_zt));

   CHECK_ERROR(hipEventRecord(stop, 0));
   CHECK_ERROR(hipEventSynchronize(stop));
   float elapsedTime;
   CHECK_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
   CHECK_ERROR(hipEventDestroy(start));
   CHECK_ERROR(hipEventDestroy(stop));
   std::cout << "CUDA elapsed: " << elapsedTime / 1000.0 << std::endl;

   delete [] xt;
   delete [] yt;
   delete [] zt;

   int ***grid;
   grid = new int **[maxdim];
   for (int i = 0; i < maxdim; ++i) {
      grid[i] = new int *[maxdim];
      for (int j = 0; j < maxdim; ++j) {
         grid[i][j] = new int[maxdim];
         for (int k = 0; k < maxdim; ++k)
            grid[i][j][k] = 0;
      }
   }
   std::cout << *(x + 0) << " " << *(y + 0) << " " << *(z + 0) << std::endl;
   for (int i = 0; i < readnum; ++i) {
      grid[*(x + i)][*(y + i)][*(z + i)] = i;
   }

   std::cout << x[0] << " : " << y[0] << " : " << z[0] << std::endl;

   collision(x, y, z, vx, vy, vz, readnum, maxdim, grid, timestep * stepnum, ofresult);

   t = clock() - t;
   double seconds = (double)t / CLOCKS_PER_SEC;

   std::cout << std::endl << "Total time consumed: " << seconds << " seconds" << std::endl;
   std::cout << "Result output to file: " << ofs_result << std::endl;

   std::cout << std::endl << "************Config Info*************" << std::endl;
   std::cout << " Particle Num: " << particle_num << std::endl;
   std::cout << "    Time step: " << timestep << std::endl;
   std::cout << "      Max dim: -" << maxdim << " ~ +" << maxdim << std::endl;
   std::cout << "Time step num: " << stepnum << std::endl;
   std::cout << std::endl << "************End*************" << std::endl;

   ofresult << std::endl << "Total time consumed: " << seconds << " seconds" << std::endl;
   ofresult << std::endl << "************Config Info*************" << std::endl;
   ofresult << " Particle Num: " << particle_num << std::endl;
   ofresult << "    Time step: " << timestep << std::endl;
   ofresult << "      Max dim: -" << maxdim << " ~ +" << maxdim << std::endl;
   ofresult << "Time step num: " << stepnum << std::endl;
   ofresult << std::endl << "************End*************" << std::endl;

   ofresult.close();

   delete [] x;
   delete [] y;
   delete [] z;
   delete [] vx;
   delete [] vy;
   delete [] vz;
   for (int i = 0; i < maxdim; ++i) {
      for (int j = 0; j < maxdim; ++j) {
         delete[] grid[i][j];
      }
      delete[] grid[i];
   }
   delete[] grid;
   std::cout << std::endl;
   return 0;
}

void init(std::vector<double *> &ppv, const std::vector<Particle> &pv)
{
   int readnum = pv.size();
   for (int i = 0; i < readnum; ++i) {
      *(ppv[0] + i) = (pv[i]).xyz.x;
      *(ppv[1] + i) = (pv[i]).xyz.y;
      *(ppv[2] + i) = (pv[i]).xyz.z;
      *(ppv[3] + i) = (pv[i]).v.x;
      *(ppv[4] + i) = (pv[i]).v.y;
      *(ppv[5] + i) = (pv[i]).v.z;
   }
}

double scalev(double &num, const double &factor)
{
   return (num + num * factor);
}

void swapv(double *vx, double *vy, double *vz, int num1, int num2, double factor)
{
   double tvx, tvy, tvz;
   if (vx[num1] + vy[num1] + vz[num1] < 1.0) {
      tvx = scalev(vx[num1], factor);
      tvy = scalev(vy[num1], factor);
      tvz = scalev(vz[num1], factor);
   }
   else {
      tvx = vx[num1];
      tvy = vy[num1];
      tvz = vz[num1];
   }
   if (vx[num2] + vy[num2] + vz[num2] < 1.0) {
      vx[num1] = scalev(vx[num2], factor);
      vy[num1] = scalev(vy[num2], factor);
      vz[num1] = scalev(vz[num2], factor);
   }
   else {
      vx[num1] = vx[num2];
      vy[num1] = vy[num2];
      vz[num1] = vz[num2];
   }
   vx[num2] = tvx;
   vy[num2] = tvy;
   vz[num2] = tvz;
}

bool isInGrid(const int &x, const int &y, const int &z, const int &gdim)
{
   return !( x > gdim || y > gdim || z > gdim);
}

unsigned updatePosition(unsigned *x, unsigned *y, unsigned *z, double *vx, double *vy, double *vz, const int &num, const int &gdim, int ***grid, const unsigned long &time)
{
   double fix_step_length = 2.0;
   double fix_speed = 0.2;
   double fix_hit_v = 0.2;
   while ((fabs(vx[num]) + fabs(vy[num]) + fabs(vz[num])) * fix_step_length < 1.0)
      fix_step_length += 2.0;
   unsigned hit_num = 0;
   unsigned long ttime = time;
   if (!isInGrid(x[num], y[num], z[num], gdim))
      runError("Particle out of bound", "update_position");
   while (ttime--) {
      int tx = x[num];
      int ty = y[num];
      int tz = z[num];
      int fx = rint(vx[num] * fix_step_length);
      int fy = rint(vy[num] * fix_step_length);
      int fz = rint(vz[num] * fix_step_length);

      tx += ((fx < 0 && abs(fx) > tx) ? 0 : fx);
      ty += ((fy < 0 && abs(fy) > ty) ? 0 : fy);
      tz += ((fz < 0 && abs(fz) > tz) ? 0 : fz);

      if (tx >= gdim || tx < 0) {
         ++hit_num;
         tx -= ((fx < 0 && abs(fx) > tx) ? 0 : fx);
         if (tx >= gdim || tx < 0) 
            tx %= (gdim - 1);
         if (vy[num] < 1.0 || vz[num] < 1.0) {
            vy[num] += vx[num] * fix_speed;
            vz[num] += vx[num] * fix_speed;
         }
         vx[num] = - vx[num];
      }
      if (ty >= gdim || ty < 0) {
         ++hit_num;
         ty -= ((fy < 0 && abs(fy) > ty) ? 0 : fy);
         if (ty >= gdim || ty < 0) 
            ty %= (gdim - 1);
         if (vx[num] < 1.0 || vz[num] < 1.0){
            vx[num] += vy[num] * fix_speed;
            vz[num] += vy[num] * fix_speed;
         } 
         vy[num] = - vy[num];
      }
      if (tz >= gdim || tz < 0) {
         ++hit_num;
         tz -= ((fz < 0 && abs(fz) > tz) ? 0 : fz);
         if (tz >= gdim || tz < 0) 
            tz %= (gdim - 1);
         if (vx[num] < 1.0 || vy[num] < 1.0) {
            vy[num] += vz[num] * fix_speed;
            vx[num] += vz[num] * fix_speed;
         }
         vz[num] = - vz[num];
      }
      if (grid[tx][ty][tz] == 0) {
         grid[tx][ty][tz] = num;
         grid[x[num]][y[num]][z[num]] = 0;
         x[num] = tx;
         y[num] = ty;
         z[num] = tz;
      }
      else {
         ++hit_num;
         int tn = grid[tx][ty][tz];
         swapv(vx, vy, vz, num, tn, fix_hit_v);
         grid[x[num]][y[num]][z[num]] = 0;
         x[num] = tx;
         y[num] = ty;
         z[num] = tz;
         grid[tx][ty][tz] = num;
         while (grid[tx][ty][tz] != 0) {
            int tx_old = tx;
            int ty_old = ty;
            int tz_old = tz;
            int fx = rint(vx[tn] * fix_step_length);
            int fy = rint(vy[tn] * fix_step_length);
            int fz = rint(vz[tn] * fix_step_length);

            tx += ((fx < 0 && abs(fx) > tx) ? 0 : fx);
            ty += ((fy < 0 && abs(fy) > ty) ? 0 : fy);
            tz += ((fz < 0 && abs(fz) > tz) ? 0 : fz);

            if (tx >= gdim || tx < 0) {
               ++hit_num;
               tx -= ((fx < 0 && abs(fx) > tx) ? 0 : fx);
               if (tx >= gdim || tx < 0) 
                  tx %= (gdim - 1);
               if (vy[tn] < 1.0 || vz[tn] < 1.0) {
                  vy[tn] += vx[tn] * fix_speed;
                  vz[tn] += vx[tn] * fix_speed;
               }
               vx[tn] = - vx[tn];
            }
            if (ty >= gdim || ty < 0) {
               ++hit_num;
               ty -= ((fy < 0 && abs(fy) > ty) ? 0 : fy);
               if (ty >= gdim || ty < 0) 
                  ty %= (gdim - 1);
               if (vx[tn] < 1.0 || vz[tn] < 1.0){
                  vx[tn] += vy[tn] * fix_speed;
                  vz[tn] += vy[tn] * fix_speed;
               } 
               vy[tn] = - vy[tn];
            }
            if (tz >= gdim || tz < 0) {
               ++hit_num;
               tz -= ((fz < 0 && abs(fz) > tz) ? 0 : fz);
               if (tz >= gdim || tz < 0) 
                  tz %= (gdim - 1);
               if (vx[tn] < 1.0 || vy[tn] < 1.0) {
                  vy[tn] += vz[tn] * fix_speed;
                  vx[tn] += vz[tn] * fix_speed;
               }
               vz[tn] = - vz[tn];
            }
            if (tx_old == tx || ty_old == ty || tz_old == tz)
               break;
            if (grid[tx][ty][tz] != 0) {
               int ttn = grid[tx][ty][tz];
               swapv(vx, vy, vz, tn, ttn, fix_hit_v);
               ++hit_num;
            }
         }
         grid[tx][ty][tz] = tn;
      }
   }
   return hit_num;
}

unsigned long long collision(unsigned *x, unsigned *y, unsigned *z, double *vx, double *vy, double *vz, const int &readnum, const int &gdim, int ***grid, const unsigned long &time, std::ostream &os)
{
   unsigned long long total_hit = 0;
   for (int i = 0; i < readnum; ++i) {
      unsigned hit_times = updatePosition(x, y, z, vx, vy, vz, i, maxdim, grid, time);
      total_hit += hit_times;
      std::cout << std::endl << "Particle " << i + 1 << " hit times: " << hit_times << std::endl;
      std::cout << "      Total hit times: " << total_hit << std::endl;
      std::cout << "Particle current info: " << std::endl;

      os.setf(std::ios::scientific);
      os.precision(19);
      os << std::endl << "********************" << std::endl;
      os << "Particle " << i + 1 << " hit times: " << hit_times << std::endl;
      os << "Particle origin info: " << std::endl;
      os << "Particle current info: " << std::endl;
      os << "Total hit times: " << total_hit << std::endl << std::endl;
   }
   return total_hit;
}

